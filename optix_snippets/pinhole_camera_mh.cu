#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2015 Jefferson Amstutz                                         //
// Copyright 2015 SURVICE Engineering Company                               //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include <optix_world.h>
#include "helpers.h"

using namespace optix;

#include "RayPayload.h"

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float3, bad_color, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(unsigned int, multihit_ray_type, , );
rtBuffer<uchar4, 2> output_buffer;
#if MHTK_SOA_LAYOUT
rtBuffer<float, 3> hits_T;
rtBuffer<int,   3> hits_PrimID;
rtBuffer<int,   3> hits_GeomID;
rtBuffer<float, 3> hits_Ngx;
rtBuffer<float, 3> hits_Ngy;
rtBuffer<float, 3> hits_Ngz;
#else
rtBuffer<Hitpoint, 3> hits_buffer;
#endif

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(float, time_view_scale, , ) = 1e-6f;

static __host__ __device__ __inline__
optix::float3 make_grad_color(const int i, const int max)
{
  float f = i;
  if (i == 0)
    return make_float3(0.f);                                   // black
  else if (i <= max/4)
    return make_float3(0.f, 0.f, f/(max/4));                   // bluish
  else if (i <= max/2)
    return make_float3(0.f, f/(max/4) - 1.f, 2.f - f/(max/4)); // cyan/green
  else if (i <= 3*max/4)
    return make_float3(f/(max/4)-2.f, 1.f, 0.f);               // green/yellow
  else if (i <= max)
    return make_float3(1.f, 4.0-f/(max/4), 0.f);               // orange
  else
    return make_float3(1.f, 0.0, 0.f);                         // red
}


RT_PROGRAM void pinhole_camera_mh()
{
  float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W);

  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, multihit_ray_type,
                                   scene_epsilon, RT_DEFAULT_MAX);

  PerRayData_multihit prd;
  prd.hitBufferOverflow = false;
  prd.numHits  = 0;
  prd.numSwaps = 0;

  rtTrace(top_object, ray, prd);

#if MHTK_SORTING
# if MHTK_POST_SORTING
#  if MHTK_SOA_LAYOUT
  if (prd.numHits > 0)
  {
#    if REVERSED_ACCESS
    uint3 i = make_uint3(0u, launch_index.y, launch_index.x);
    for (; i.x < prd.numHits - 1; ++(i.x))
    {
#    else
    uint3 i = make_uint3(launch_index.x, launch_index.y, 0u);
    for (; i.z < prd.numHits - 1; ++(i.z))
    {
#    endif
      float d = hits_T[i];
      uint3 pos = i;
      bool doSwap = false;
#    if REVERSED_ACCESS
      uint3 j = make_uint3(i.x+1, i.y, i.z);
      for (; j.x < prd.numHits; ++(j.x))
      {
#    else
      uint3 j = make_uint3(i.x, i.y, i.z+1);
      for (; j.z < prd.numHits; ++(j.z))
      {
#    endif
        if(hits_T[j] < d)
        {
          doSwap = true;
          pos = j;
          d = hits_T[j];
        }
      }

      if (doSwap)
      {
        Hitpoint tmp;
        tmp.t      = hits_T[i];
        tmp.primID = hits_PrimID[i];
        tmp.geomID = hits_GeomID[i];
        tmp.Ng[0]  = hits_Ngx[i];
        tmp.Ng[1]  = hits_Ngy[i];
        tmp.Ng[2]  = hits_Ngz[i];

        hits_T[i]      = hits_T[pos];
        hits_PrimID[i] = hits_PrimID[pos];
        hits_GeomID[i] = hits_GeomID[pos];
        hits_Ngx[i]    = hits_Ngx[pos];
        hits_Ngy[i]    = hits_Ngy[pos];
        hits_Ngz[i]    = hits_Ngz[pos];

        hits_T[pos]      = tmp.t;
        hits_PrimID[pos] = tmp.primID;
        hits_GeomID[pos] = tmp.geomID;
        hits_Ngx[pos]    = tmp.Ng[0];
        hits_Ngy[pos]    = tmp.Ng[1];
        hits_Ngz[pos]    = tmp.Ng[2];
#if VIZ_DATA
        prd.numSwaps++;
#endif
      }
    }
  }
#  else
  if (prd.numHits > 0)
  {
    uint3 i = make_uint3(launch_index.x, launch_index.y, 0u);
    /* sort the hitpoints (only for measuring performance) */
    for (; i.z < prd.numHits - 1; ++(i.z))
    {
      float d = hits_buffer[i].t;
      uint3 pos = i;
      bool doSwap = false;
      uint3 j = make_uint3(i.x, i.y, i.z+1);
      for (; j.z < prd.numHits; ++(j.z))
      {
        if(hits_buffer[j].t < d)
        {
          doSwap = true;
          pos = j;
          d = hits_buffer[j].t;
        }
      }

      if (doSwap)
      {
        Hitpoint tmp     = hits_buffer[i];
        hits_buffer[i]   = hits_buffer[pos];
        hits_buffer[pos] = tmp;
#if VIZ_DATA
        prd.numSwaps++;
#endif
      }
    }
  }
#  endif
# endif
#endif

#if VIZ_DATA
# if VIZ_SWAPS
  output_buffer[launch_index] = make_color(make_grad_color(prd.numSwaps, 128));
# else
  output_buffer[launch_index] = make_color(make_grad_color(prd.numHits, 128));
# endif
#else
  if (!prd.hitBufferOverflow)
    output_buffer[launch_index] = make_color(make_float3(prd.numHits / 10.f));
  else
    output_buffer[launch_index] = make_color(make_float3(1.f, 0.f, 0.f));
#endif
}

RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
  output_buffer[launch_index] = make_color( bad_color );
}
