#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2015 Jefferson Amstutz                                         //
// Copyright 2015 SURVICE Engineering Company                               //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include <optix_world.h>
#include "random.h"
#include "commonStructs.h"

#include "RayPayload.h"

using namespace optix;

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtDeclareVariable(PerRayData_multihit, prd_mh, rtPayload, );

#if MHTK_SOA_LAYOUT
rtBuffer<float, 3> hits_T;
rtBuffer<int,   3> hits_PrimID;
rtBuffer<int,   3> hits_GeomID;
rtBuffer<float, 3> hits_Ngx;
rtBuffer<float, 3> hits_Ngy;
rtBuffer<float, 3> hits_Ngz;
#else
rtBuffer<Hitpoint, 3> hits_buffer;
#endif
RT_PROGRAM void any_hit_multihit()
{
  if (prd_mh.numHits >= HITPOINT_BUFFER_SIZE)
  {
    prd_mh.hitBufferOverflow = true;
    rtTerminateRay();
    return;
  }

#if MHTK_SORTING
# if !MHTK_POST_SORTING
#  if MHTK_SOA_LAYOUT
#   if REVERSED_ACCESS
  uint3 i = make_uint3(prd_mh.numHits, launch_index.y, launch_index.x);
#   else
  uint3 i = make_uint3(launch_index.x, launch_index.y, prd_mh.numHits);
#   endif

#   if REVERSED_ACCESS
  for (; i.x > 0; --i.x)
  {
    uint3 j = make_uint3(i.x-1, i.y, i.z);
#   else
  for (; i.z > 0; --i.z)
  {
    uint3 j = make_uint3(i.x, i.y, i.z-1);
#   endif
    if (hits_T[j] > t_hit)
    {
      hits_T[i]      = hits_T[j];
      hits_PrimID[i] = hits_PrimID[j];
      hits_GeomID[i] = hits_GeomID[j];
      hits_Ngx[i]    = hits_Ngx[j];
      hits_Ngy[i]    = hits_Ngy[j];
      hits_Ngz[i]    = hits_Ngz[j];
#   if VIZ_DATA
      prd_mh.numSwaps++;
#   endif
    }
    else
      break;
  }

  hits_T[i]      = t_hit;
  hits_Ngx[i]    = geometric_normal.x;
  hits_Ngy[i]    = geometric_normal.y;
  hits_Ngz[i]    = geometric_normal.z;
  hits_PrimID[i] = 0;
  hits_PrimID[i] = 0;
  prd_mh.numHits++;
#  else
  uint3 i = make_uint3(launch_index.x, launch_index.y, prd_mh.numHits);

  for (; i.z > 0; --i.z)
  {
    uint3 j = make_uint3(i.x, i.y, i.z-1);
    if (hits_buffer[j].t > t_hit)
    {
      hits_buffer[i] = hits_buffer[j];
#   if VIZ_DATA
      prd_mh.numSwaps++;
#   endif
    }
    else
      break;
  }

  Hitpoint &hit = hits_buffer[i];
  hit.t      = t_hit;
  hit.Ng[0]  = geometric_normal.x;
  hit.Ng[1]  = geometric_normal.y;
  hit.Ng[2]  = geometric_normal.z;
  hit.primID = 0;
  hit.geomID = 0;
  prd_mh.numHits++;
#  endif
# else
#  if MHTK_SOA_LAYOUT
#    if REVERSED_ACCESS
  uint3 index = make_uint3(prd_mh.numHits++, launch_index.y, launch_index.x);
#    else
  uint3 index = make_uint3(launch_index.x, launch_index.y, prd_mh.numHits++);
#    endif
  hits_T[index]      = t_hit;
  hits_Ngx[index]    = geometric_normal.x;
  hits_Ngy[index]    = geometric_normal.y;
  hits_Ngz[index]    = geometric_normal.z;
  hits_PrimID[index] = 0;
  hits_GeomID[index] = 0;
#  else
#   if REVERSED_ACCESS
  uint3 index = make_uint3(prd_mh.numHits++, launch_index.y, launch_index.x);
#   else
  uint3 index = make_uint3(launch_index.x, launch_index.y, prd_mh.numHits++);
#   endif
  Hitpoint &hit = hits_buffer[index];
  hit.t = t_hit;
  hit.Ng[0] = geometric_normal.x;
  hit.Ng[1] = geometric_normal.y;
  hit.Ng[2] = geometric_normal.z;
  hit.primID = 0;
  hit.geomID = 0;
#  endif
# endif
#else
# if MHTK_SOA_LAYOUT
#  if REVERSED_ACCESS
  uint3 index = make_uint3(prd_mh.numHits++, launch_index.y, launch_index.x);
#  else
  uint3 index = make_uint3(launch_index.x, launch_index.y, prd_mh.numHits++);
#  endif
  hits_T[index]      = t_hit;
  hits_Ngx[index]    = geometric_normal.x;
  hits_Ngy[index]    = geometric_normal.y;
  hits_Ngz[index]    = geometric_normal.z;
  hits_PrimID[index] = 0;
  hits_GeomID[index] = 0;
# else
#  if REVERSED_ACCESS
  uint3 index = make_uint3(prd_mh.numHits++, launch_index.y, launch_index.x);
#  else
  uint3 index = make_uint3(launch_index.x, launch_index.y, prd_mh.numHits++);
#  endif
  Hitpoint &hit = hits_buffer[index];
  hit.t = t_hit;
  hit.Ng[0] = geometric_normal.x;
  hit.Ng[1] = geometric_normal.y;
  hit.Ng[2] = geometric_normal.z;
  hit.primID = 0;
  hit.geomID = 0;
# endif
#endif

  rtIgnoreIntersection();
}

// ----------------------------------------------------------


